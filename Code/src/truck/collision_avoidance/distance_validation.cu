#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include <stdbool.h>
#include<hip/hip_runtime_api.h>

using namespace std;
#define Number_of_sensor 4

// Kernel
__global__ void distance_validation(double *sensor_input, double *treshold, double *output)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    output[i] =    sensor_input[i] - treshold[i];
}


int main(){


    //Vector Declaration and Definition
    double sensor_input[Number_of_sensor] =   { 9.0  , 8.0 , 0.5 , 6.0   };
    double treshold[Number_of_sensor]     =   { 3.0  , 3.0 , 0.8 , 0.8  };
    double output[Number_of_sensor]       =   { 0.0  , 0.0 , 0.0 , 0.0  };
    double *d_sensor_input, *d_treshold, *d_output;


    //Allocation of Device variables
    hipMalloc((void **)&d_sensor_input, sizeof(double)*Number_of_sensor);
    hipMalloc((void **)&d_treshold, sizeof(double)*Number_of_sensor);
    hipMalloc((void **)&d_output, sizeof(double)*Number_of_sensor);

    //Copy Host Memory to Device Memory
    hipMemcpy(d_sensor_input, &sensor_input, sizeof(double)*Number_of_sensor, hipMemcpyHostToDevice);
    hipMemcpy(d_treshold, &treshold, sizeof(double)*Number_of_sensor, hipMemcpyHostToDevice);
    hipMemcpy(d_output, &output, sizeof(double)*Number_of_sensor, hipMemcpyHostToDevice);

    int thr_per_blk = 3;
    int blk_in_grid = ceil( float(Number_of_sensor) / thr_per_blk );


    // Launch kernel
    distance_validation<<< blk_in_grid, thr_per_blk >>>(d_sensor_input, d_treshold, d_output);

     //Copy Device Memory to Host Memory
    hipMemcpy(&output, d_output, sizeof(double)*Number_of_sensor, hipMemcpyDeviceToHost);


    // print result



    printf("Sensor position: front  \t  back  \t left  \t\t  right  \t\n\n");

    printf("Sensor input:  \t %f \t %f \t %f \t %f \t\n\n", sensor_input[0], sensor_input[1], sensor_input[2], sensor_input[3]);

    printf("treshold:  \t %f \t %f \t %f \t %f \t\n\n", treshold[0], treshold[1], treshold[2], treshold[3]);

    printf("output:  \t %f \t %f \t %f \t %f \t\n\n", output[0], output[1], output[2], output[3]);

    std::string v_1 = (output[0]>0)?"true":"false";
    std::string v_2 = (output[1]>0)?"true":"false";
    std::string v_3 = (output[2]>0)?"true":"false";
    std::string v_4 = (output[3]>0)?"true":"false";

    std::cout << "validation: "<< "  \t " << v_1 << "\t\t  " << v_2 << "\t\t " << v_3 << "\t\t " << v_4 << "\n\n" << std::endl;



    //front sensor
    if(output[0]<0){
      printf("front side is to close with obstacle !!!!\n");
    }

    //back sensor
    if(output[1]<0){
      printf("back side is to close with obstacle !!!!\n");
    }

    //left sensor
    if(output[2]<0){
      printf("left side is to close with obstacle !!!!\n");
    }

    //right sensor
    if(output[3]<0){
      printf("right side is to close with obstacle !!!!\n");
    }


    // Free GPU memory
    hipFree(d_sensor_input);
    hipFree(d_treshold);
    hipFree(d_output);

    return 0;
}
